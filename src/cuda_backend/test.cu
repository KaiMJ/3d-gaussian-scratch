#include <hip/hip_runtime.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void add(int* c, const int* a, const int* b, int size) {
    int* d_a;
    int* d_b;
    int* d_c;

    hipMalloc(&d_a, size * sizeof(int));
    hipMalloc(&d_b, size * sizeof(int));
    hipMalloc(&d_c, size * sizeof(int));

    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    addKernel<<<numBlocks, blockSize>>>(d_c, d_a, d_b, size);

    hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}